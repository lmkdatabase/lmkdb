#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <fstream>
#include <sstream>
#include <unordered_map>
#include <string>
#include <hip/hip_runtime.h>
#include <filesystem>
#include <thread>

using namespace std;
namespace fs = std::filesystem;

#define CUDA_CALL(call)                                                      \
    {                                                                        \
        hipError_t err = call;                                              \
        if (err != hipSuccess) {                                            \
            cerr << "CUDA Error at " << __FILE__ << ":" << __LINE__ << " : " \
                 << hipGetErrorString(err) << endl;                         \
            exit(EXIT_FAILURE);                                              \
        }                                                                    \
    }

const int MAX_STRING_LEN = 256;
const int MAX_BATCH_SIZE = 10000;

struct JoinTask {
    string tb1File;
    string tb2File;
    string outputFile;
    string metadata1;
    string metadata2;
};

vector<string> getShardPaths(const string& folderPath) {
    vector<string> shardPaths;

    for (const auto& entry : fs::directory_iterator(folderPath)) {
        if (entry.is_regular_file() && entry.path().extension() == ".csv") {
            shardPaths.push_back(entry.path().string());
        }
    }

    return shardPaths;
}

unordered_map<string, int> loadMetadata(const string& metadataFile) {
    unordered_map<string, int> metadata;
    ifstream file(metadataFile);
    if (!file.is_open()) {
        cerr << "Error: Could not open metadata file " << metadataFile << endl;
        exit(EXIT_FAILURE);
    }

    string line;
    while (getline(file, line)) {
        istringstream ss(line);
        string attribute;
        int index;
        if (getline(ss, attribute, ',') && ss >> index) {
            metadata[attribute] = index;
        }
    }
    return metadata;
}

__device__ int device_strcmp(const char* s1, const char* s2) {
    while (*s1 && (*s1 == *s2)) {
        s1++;
        s2++;
    }
    return *(const unsigned char*)s1 - *(const unsigned char*)s2;
}

__device__ void device_strcpy(char* dest, const char* src) {
    while ((*dest++ = *src++) != '\0');
}

__global__ void joinKernel(char* tb1, char* tb2, char* results, int* resultCount,
                           int cols1, int cols2, int size1, int size2,
                           int joinIdx1, int joinIdx2, int maxResults) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= size2) return;

    for (int i = 0; i < size1; i++) {
        if (device_strcmp(&tb1[i * cols1 * MAX_STRING_LEN + joinIdx1 * MAX_STRING_LEN],
                          &tb2[idx * cols2 * MAX_STRING_LEN + joinIdx2 * MAX_STRING_LEN]) == 0) {
            int insertIdx = atomicAdd(resultCount, 1);

            if (insertIdx < maxResults) {
                for (int j = 0; j < cols1; j++) {
                    device_strcpy(&results[insertIdx * (cols1 + cols2) * MAX_STRING_LEN + j * MAX_STRING_LEN],
                                  &tb1[i * cols1 * MAX_STRING_LEN + j * MAX_STRING_LEN]);
                }
                for (int j = 0; j < cols2; j++) {
                    device_strcpy(&results[insertIdx * (cols1 + cols2) * MAX_STRING_LEN +
                                           (cols1 + j) * MAX_STRING_LEN],
                                  &tb2[idx * cols2 * MAX_STRING_LEN + j * MAX_STRING_LEN]);
                }
            } else {
                return;
            }
        }
    }
}

size_t readBatch(ifstream& file, char* buffer, int numCols) {
    size_t currSize = 0;
    string line;

    while (currSize < MAX_BATCH_SIZE && getline(file, line)) {
        istringstream ss(line);
        string value;
        int col = 0;

        while (col < numCols && getline(ss, value, ',')) {
            strncpy(&buffer[currSize * numCols * MAX_STRING_LEN + col * MAX_STRING_LEN],
                    value.c_str(), MAX_STRING_LEN - 1);
            buffer[currSize * numCols * MAX_STRING_LEN + col * MAX_STRING_LEN + MAX_STRING_LEN - 1] = '\0';
            col++;
        }
        currSize++;
    }

    return currSize;
}

void processPairOnGPU(const JoinTask& jTask, const string& joinAttr) {
    auto metadata1 = loadMetadata(jTask.metadata1);
    auto metadata2 = loadMetadata(jTask.metadata2);

    if (metadata1.find(joinAttr) == metadata1.end() ||
        metadata2.find(joinAttr) == metadata2.end()) {
        cerr << "Error: Join attribute not found in metadata" << endl;
        return;
    }

    int joinIdx1 = metadata1[joinAttr];
    int joinIdx2 = metadata2[joinAttr];

    int cols1 = metadata1.size();
    int cols2 = metadata2.size();

    ifstream file1(jTask.tb1File);
    ifstream file2(jTask.tb2File);

    if (!file1.is_open() || !file2.is_open()) {
        cerr << "Error: Could not open input files" << endl;
        return;
    }

    ofstream outFile(jTask.outputFile, ios::out | ios::trunc);
    if (!outFile.is_open()) {
        cerr << "Error: Could not open output file" << endl;
        return;
    }

    char *batch1, *batch2;
    CUDA_CALL(hipHostAlloc(&batch1, MAX_BATCH_SIZE * cols1 * MAX_STRING_LEN, hipHostMallocDefault));
    CUDA_CALL(hipHostAlloc(&batch2, MAX_BATCH_SIZE * cols2 * MAX_STRING_LEN, hipHostMallocDefault));

    char *d_tb1, *d_tb2, *d_results;
    int *d_resultCount;
    CUDA_CALL(hipMalloc(&d_tb1, MAX_BATCH_SIZE * cols1 * MAX_STRING_LEN));
    CUDA_CALL(hipMalloc(&d_tb2, MAX_BATCH_SIZE * cols2 * MAX_STRING_LEN));
    CUDA_CALL(hipMalloc(&d_resultCount, sizeof(int)));

    hipStream_t stream;
    CUDA_CALL(hipStreamCreate(&stream));

    size_t batch1Size = 0;
    while ((batch1Size = readBatch(file1, batch1, cols1)) > 0) {
        CUDA_CALL(hipMemcpyAsync(d_tb1, batch1, batch1Size * cols1 * MAX_STRING_LEN, hipMemcpyHostToDevice, stream));

        file2.clear();
        file2.seekg(0, ios::beg);

        size_t batch2Size = 0;
        while ((batch2Size = readBatch(file2, batch2, cols2)) > 0) {
            CUDA_CALL(hipMemcpyAsync(d_tb2, batch2, batch2Size * cols2 * MAX_STRING_LEN, hipMemcpyHostToDevice, stream));

            int maxResLen = batch1Size * batch2Size;
            CUDA_CALL(hipMalloc(&d_results, maxResLen * (cols1 + cols2) * MAX_STRING_LEN));

            vector<char> results(maxResLen * (cols1 + cols2) * MAX_STRING_LEN);
            int zero = 0;
            CUDA_CALL(hipMemcpyAsync(d_resultCount, &zero, sizeof(int), hipMemcpyHostToDevice, stream));

            joinKernel<<<(batch2Size + 127) / 128, 128, 0, stream>>>(
                d_tb1, d_tb2, d_results, d_resultCount,
                cols1, cols2, batch1Size, batch2Size,
                joinIdx1, joinIdx2, maxResLen);

            CUDA_CALL(hipStreamSynchronize(stream)); 
            int resultCount = 0;
            CUDA_CALL(hipMemcpyAsync(&resultCount, d_resultCount, sizeof(int), hipMemcpyDeviceToHost, stream));
            CUDA_CALL(hipMemcpyAsync(results.data(), d_results, resultCount * (cols1 + cols2) * MAX_STRING_LEN, hipMemcpyDeviceToHost, stream));

            CUDA_CALL(hipStreamSynchronize(stream));

            for (int i = 0; i < resultCount; i++) {
                for (int j = 0; j < cols1 + cols2; j++) {
                    outFile << &results[i * (cols1 + cols2) * MAX_STRING_LEN + j * MAX_STRING_LEN];
                    if (j < cols1 + cols2 - 1) outFile << ",";
                }
                outFile << endl;
            }

            CUDA_CALL(hipFree(d_results));
        }
    }

    CUDA_CALL(hipFree(d_tb1));
    CUDA_CALL(hipFree(d_tb2));
    CUDA_CALL(hipFree(d_resultCount));
    CUDA_CALL(hipHostFree(batch1));
    CUDA_CALL(hipHostFree(batch2));
    CUDA_CALL(hipStreamDestroy(stream));
}

void processJoinTasksSubset(const vector<JoinTask>& jTasksSubset, const string& joinAttr) {
    for (const auto& jTask : jTasksSubset) {
        processPairOnGPU(jTask, joinAttr);
    }
}

void processJoinTasksParallel(const vector<jTask>& jTasks, const string& joinAttr, int numThreads) {
    vector<thread> threads;
    size_t jTasksPerThread = jTasks.size() / numThreads;

    for (int i = 0; i < numThreads; i++) {
        size_t startIdx = i * jTasksPerThread;
        size_t endIdx = (i == numThreads - 1) ? jTasks.size() : (i + 1) * jTasksPerThread;

        threads.emplace_back(processJoinTasksSubset, vector<jTask>(jTasks.begin() + startIdx, jTasks.begin() + endIdx), joinAttr);
    }

    for (auto& t : threads) {
        if (t.joinable()) {
            t.join();
        }
    }
}


int main() {
    string tb1Folder = "/content/drive/My Drive/lmkdb/london";
    string tb2Folder = "/content/drive/My Drive/lmkdb/stations";

    vector<string> tb1Shards = getShardPaths(tb1Folder);
    vector<string> tb2Shards = getShardPaths(tb2Folder);

    vector<JoinTask> jTasks;

    int outputFileIndex = 1;
    for (const string& tb1File : tb1Shards) {
        for (const string& tb2File : tb2Shards) {
            JoinTask jTask;
            jTask.tb1File = tb1File;
            jTask.tb2File = tb2File;

            jTask.outputFile = "res_" + to_string(outputFileIndex++) + ".csv";

            jTask.metadata1 = tb1Folder + "/metadata.txt";
            jTask.metadata2 = tb2Folder + "/metadata.txt";

            jTasks.push_back(jTask);
        }
    }

    string joinAttr = "start_station_id";

    processJoinTasksParallel(jTasks, joinAttr, 2);
    return 0;
}
