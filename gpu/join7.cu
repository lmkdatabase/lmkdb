#include "hip/hip_runtime.h"
#include <fstream>
#include <unordered_map>
#include <vector>
#include <string>
#include <sstream>
#include <filesystem>
#include "hip/hip_runtime.h"
#include ""
#include "thrust/device_vector.h"
#include "thrust/host_vector.h"
#include <memory>
#include <iostream>
#include <chrono>

// Utility function to split strings (CSV parsing)
__host__ std::vector<std::string> split(const std::string& str, char delimiter) {
    std::vector<std::string> tokens;
    std::stringstream ss(str);
    std::string token;
    while (std::getline(ss, token, delimiter)) {
        tokens.push_back(token);
    }
    return tokens;
}

// Load a CSV file and extract rows and a specific column
__host__ void loadCsv(const std::string& filename, int columnIndex,
             std::vector<std::string>& rows, std::vector<int>& column,
             std::unordered_map<std::string, int>& stringToIntMap,
             std::unordered_map<int, std::string>& intToStringMap) {
    std::ifstream file(filename);
    std::string line;
    int currentIndex = 0;

    while (std::getline(file, line)) {
        rows.push_back(line); // Save the full row
        std::vector<std::string> tokens = split(line, ',');
        if (columnIndex < tokens.size()) {
            const std::string& value = tokens[columnIndex];
            if (stringToIntMap.find(value) == stringToIntMap.end()) {
                stringToIntMap[value] = currentIndex;
                intToStringMap[currentIndex] = value; // Reverse mapping
                ++currentIndex;
            }
            column.push_back(stringToIntMap[value]); // Save mapped column value
        }
    }
}

// GPU join kernel to find match indices
__global__ void joinWithIndices(const int* const left, const int* const right, int leftSize, int rightSize, int* matches) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < leftSize) {
        for (int j = 0; j < rightSize; ++j) {
            if (left[idx] == right[j]) {
                matches[idx] = j; 
                return; 
            }
        }
        matches[idx] = -1;
    }
}

// GPU join execution
std::unique_ptr<thrust::host_vector<int>> runGpuJoinWithIndices(const thrust::device_vector<int>& left,
                                                                const thrust::device_vector<int>& right) {
    int leftSize = left.size();
    thrust::device_vector<int> matches(leftSize, -1);

    int threadCount = 1024;
    int nBlocks = (leftSize + threadCount - 1) / threadCount;

    joinWithIndices<<<nBlocks, threadCount>>>(thrust::raw_pointer_cast(left.data()),
                                              thrust::raw_pointer_cast(right.data()),
                                              leftSize, right.size(),
                                              thrust::raw_pointer_cast(matches.data()));

    hipDeviceSynchronize();

    return std::make_unique<thrust::host_vector<int>>(matches);
}

__host__ std::vector<std::string> getShardPaths(const std::string& shardDir) {
    std::vector<std::string> shardPaths;
    for (const auto& entry : std::filesystem::directory_iterator(shardDir)) {
        shardPaths.push_back(entry.path().string());
    }
    return shardPaths;
}

int main() {
    auto start = std::chrono::high_resolution_clock::now();
    
    std::vector<std::string> leftShards = getShardPaths("/content/drive/My Drive/lmkdb/london2");
    std::vector<std::string> rightShards = getShardPaths("/content/drive/My Drive/lmkdb/stations/");

    auto shardLoadEnd = std::chrono::high_resolution_clock::now();
    std::cout << "Shard paths loaded in " 
              << std::chrono::duration_cast<std::chrono::milliseconds>(shardLoadEnd - start).count()
              << " ms.\n";

    std::ofstream outputFile("joined_table.csv", std::ios::app);
    if (!outputFile.is_open()) {
        std::cerr << "Error: Could not open output file.\n";
        return 1;
    }

    for (const auto& leftShard : leftShards) {
        for (const auto& rightShard : rightShards) {
            std::cout << "Processing shards: " << leftShard << " and " << rightShard << std::endl;

            // Start timing for loading CSV files
            auto loadStart = std::chrono::high_resolution_clock::now();

            std::vector<std::string> leftRows, rightRows;
            std::vector<int> leftColumn, rightColumn;
            std::unordered_map<std::string, int> stringToIntMap;
            std::unordered_map<int, std::string> intToStringMap;

            loadCsv(leftShard, 5, leftRows, leftColumn, stringToIntMap, intToStringMap);
            loadCsv(rightShard, 1, rightRows, rightColumn, stringToIntMap, intToStringMap);

            auto loadEnd = std::chrono::high_resolution_clock::now();
            std::cout << "CSV files loaded in " 
                      << std::chrono::duration_cast<std::chrono::milliseconds>(loadEnd - loadStart).count()
                      << " ms.\n";

            // Start timing for GPU operations
            // auto gpuStart = std::chrono::high_resolution_clock::now();

            // thrust::device_vector<int> leftGpu(leftColumn.begin(), leftColumn.end());
            // thrust::device_vector<int> rightGpu(rightColumn.begin(), rightColumn.end());

            // auto matches = runGpuJoinWithIndices(leftGpu, rightGpu);

            // auto gpuEnd = std::chrono::high_resolution_clock::now();
            // std::cout << "GPU operations completed in " 
            //           << std::chrono::duration_cast<std::chrono::milliseconds>(gpuEnd - gpuStart).count()
            //           << " ms.\n";

            // // Start timing for writing output
            // auto writeStart = std::chrono::high_resolution_clock::now();

            // for (size_t i = 0; i < matches->size(); ++i) {
            //     int matchIndex = (*matches)[i];
            //     if (matchIndex != -1) {
            //         std::string leftRow = leftRows[i];
            //         std::string rightRow = rightRows[matchIndex];
            //         outputFile << leftRow << "," << rightRow << "\n";
            //     }
            // }

            // auto writeEnd = std::chrono::high_resolution_clock::now();
            // std::cout << "Output writing completed in " 
            //           << std::chrono::duration_cast<std::chrono::milliseconds>(writeEnd - writeStart).count()
            //           << " ms.\n";
        }
    }

    outputFile.close();

    auto end = std::chrono::high_resolution_clock::now();
    std::cout << "Total execution time: " 
              << std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count()
              << " ms.\n";

    return 0;
}
