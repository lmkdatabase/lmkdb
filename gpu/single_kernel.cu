#include "hip/hip_runtime.h"
#include <fstream>
#include <unordered_map>
#include <vector>
#include <string>
#include <sstream>
#include <filesystem>
#include <memory>
#include <iostream>
#include <chrono>

#include "hip/hip_runtime.h"
#include ""
#include "thrust/device_vector.h"
#include "thrust/host_vector.h"

using namespace std;

__host__ vector<string> split(const string& str, char delimiter) {
    vector<string> tokens;
    size_t start = 0, end;

    while ((end = str.find(delimiter, start)) != string::npos) {
        tokens.push_back(str.substr(start, end - start));
        start = end + 1;
    }

    tokens.push_back(str.substr(start));
    return tokens;
}

__host__ void loadCsv(const string& filename, int joinIdx,
             vector<string>& rows, vector<int>& column,
             unordered_map<string, int>& strToIntMap,
             unordered_map<int, string>& intToStrMap) {
    ifstream file(filename);
    if (!file.is_open()) {
        cerr << "Error: Could not open file " << filename << "\n";
        return;
    }

    string line;
    int currentIndex = 0;
    int line_count = 0;

    while (getline(file, line)) {
        line_count++;
        rows.push_back(line);

        istringstream lineStream(line);
        string field;
        int colIndex = 0;
        while (getline(lineStream, field, ',')) {
            if (colIndex == joinIdx) {
                if (strToIntMap.find(field) == strToIntMap.end()) {
                    strToIntMap[field] = currentIndex;
                    intToStrMap[currentIndex] = field;
                    ++currentIndex;
                }
                column.push_back(strToIntMap[field]);
                break;
            }
            colIndex++;
        }
    }

    file.close();
}


__global__ void joinWithIndices(const int* const left, const int* const right, int leftSize, int rightSize, int* matches) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < leftSize) {
        for (int j = 0; j < rightSize; ++j) {
            if (left[idx] == right[j]) {
                matches[idx] = j; 
                return; 
            }
        }
        matches[idx] = -1;
    }
}

unique_ptr<thrust::host_vector<int>> gpuJoin(const thrust::device_vector<int>& left,
                                            const thrust::device_vector<int>& right) {
    int leftSize = left.size();
    thrust::device_vector<int> matches(leftSize, -1);

    int threadCount = 1024;
    int nBlocks = (leftSize + threadCount - 1) / threadCount;

    joinWithIndices<<<nBlocks, threadCount>>>(thrust::raw_pointer_cast(left.data()),
                                              thrust::raw_pointer_cast(right.data()),
                                              leftSize, right.size(),
                                              thrust::raw_pointer_cast(matches.data()));

    hipDeviceSynchronize();

    return make_unique<thrust::host_vector<int>>(matches);
}

__host__ vector<string> getShardPaths(const string& shardDir) {
    vector<string> shardPaths;
    for (const auto& entry : filesystem::directory_iterator(shardDir)) {
        shardPaths.push_back(entry.path().string());
    }
    return shardPaths;
}

int main() {    
    vector<string> leftShards = getShardPaths("/content/drive/My Drive/lmkdb/london");
    vector<string> rightShards = getShardPaths("/content/drive/My Drive/lmkdb/stations");

    ofstream outputFile("joined_table.csv");
    if (!outputFile.is_open()) {
        cerr << "Error: Could not open output file.\n";
        return 1;
    }

    for (const auto& leftShard : leftShards) {
        for (const auto& rightShard : rightShards) {
            vector<string> leftRows, rightRows;
            vector<int> leftColumn, rightColumn;
            
            unordered_map<string, int> strToIntMap;
            unordered_map<int, string> intToStrMap;

            loadCsv(leftShard, 5, leftRows, leftColumn, strToIntMap, intToStrMap);
            loadCsv(rightShard, 1, rightRows, rightColumn, strToIntMap, intToStrMap);

            thrust::device_vector<int> leftGpu(leftColumn.begin(), leftColumn.end());
            thrust::device_vector<int> rightGpu(rightColumn.begin(), rightColumn.end());

            auto matches = gpuJoin(leftGpu, rightGpu);

            for (size_t i = 0; i < matches->size(); ++i) {
                int matchIndex = (*matches)[i];
                if (matchIndex != -1) {
                    outputFile << leftRows[i] << "," << rightRows[matchIndex] << "\n";
                }
            }
        }
    }

    outputFile.close();
    return 0;
}
